#include "hip/hip_runtime.h"
#include<stdio.h>

#define ROW 1000
#define COL 1000

__global__ void mat_vect(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < n && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i];
        }
        c[col] = sum;
    }
}

void TestMatrixGenerate(int *matrix, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            matrix[i * col + j] = (j + 1) + i * col; 
            printf("%d ", matrix[i*col + j];)
        }
        printf("\n");
    }
}

int main() {
    int* matrix;
    TestMatrixGenerate((int*) matrix, ROW, COL);
    
    

    return 0;
}