
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ROW 1000
#define COL 1000

__global__ void mat_vect(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < n && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i];
        }
        c[col] = sum;
    }
}

void TestMatrixGenerate(int* matrix, int row, int col) {
    //printf("Row: %d", row);
    //printf("Col: %d", col);
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            matrix[i * col + j] = (j + 1) + i * col; 
      	    //printf("%d ", matrix[i*col + j]);
        }
        //printf("\n");
    }
}

void TestVectorGenerate(int* vector, int row) {
    for(int i = 0; i < row; i++) {
	vector[i] = i * 2;
 	//printf("%d ", vector[i]);
    }
}

int main() {
    int m[ROW*COL];
    int v[COL];

    int *ptr_m, *ptr_v;
    ptr_m = m;
    ptr_v = v;
    TestMatrixGenerate(ptr_m, ROW, COL);
    TestVectorGenerate(ptr_v, ROW);
    

    return 0;
}
