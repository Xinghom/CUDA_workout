#include "hip/hip_runtime.h"
#include<stdio.h>

#define ROW 1000
#define COL 1000

__global__ void mat_vect(int *a, int *b, int *c, int m, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    
    if (col < n){
        for(int i = 0; i < n; i++) 
        {
            printf("matrix value: %d, vect value: %d \n", a[row * n + i], b[i]);
            sum += a[i * m + col] * b[i];
            //printf("cur sum: %d\n", sum);
        }
	//printf("col value is : %d\n", col);
	//printf("sum value: %d\n", sum);
        c[col] = sum;
    }
}

void TestMatrixGenerate(int* matrix, int row, int col) {
    //printf("Row: %d", row);
    //printf("Col: %d", col);
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            matrix[i * col + j] = (j + 1) + i * row; 
      	    //printf("%d ", matrix[i*col + j]);
        }
        //printf("\n");
    }
}

void TestVectorGenerate(int* vector, int row) {
    for(int i = 0; i < row; i++) {
	vector[i] = i * 2;
 	//printf("%d ", vector[i]);
    }
}


int main() {
    int m[ROW*COL];
    int v[COL];
    int res[COL];

    int *ptr_m, *ptr_v, *ptr_res;
    ptr_m = m;
    ptr_v = v;
    ptr_res = res;
    TestMatrixGenerate(ptr_m, ROW, COL);
    TestVectorGenerate(ptr_v, ROW);
    
    int *d_m, *d_v;
    int *d_res;
    hipMallocManaged(&d_m, ROW*COL*sizeof(int));
    hipMallocManaged(&d_v, ROW*sizeof(int));
    hipMallocManaged(&d_res, ROW*sizeof(int));
    // init d_m, d_v on the host
    TestMatrixGenerate(d_m, ROW, COL);
    TestVectorGenerate(d_v, ROW);
    int blockSize = 256;
    mat_vect<<<COL/blockSize + 1, blockSize>>>(d_m, d_v, d_res, ROW, COL);
    
    hipDeviceSynchronize();
    
    // result verify
    printf("the front 10 elements results: \n");
    for (int i = 0; i < 10; i++) {
    	printf("%d ", d_res[i]);
    }   
    
    hipFree(d_m);
    hipFree(d_v);
    hipFree(d_res);
    return 0;
}
