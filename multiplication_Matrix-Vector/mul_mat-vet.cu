
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ROW 1000
#define COL 1000

__global__ void mat_vect(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < n && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i];
        }
        c[col] = sum;
    }
}

void TestMatrixGenerate(int* matrix, int row, int col) {
    //printf("Row: %d", row);
    //printf("Col: %d", col);
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            matrix[i * col + j] = (j + 1) + i * col; 
      	    //printf("%d ", matrix[i*col + j]);
        }
        //printf("\n");
    }
}

void TestVectorGenerate(int* vector, int row) {
    for(int i = 0; i < row; i++) {
	vector[i] = i * 2;
 	//printf("%d ", vector[i]);
    }
}

int main() {
    int m[ROW*COL];
    int v[COL];

    int *ptr_m, *ptr_v;
    ptr_m = m;
    ptr_v = v;
    TestMatrixGenerate(ptr_m, ROW, COL);
    TestVectorGenerate(ptr_v, ROW);
    
    int *d_m, *d_v;
    int *d_res;
    hipMallocManaged(&d_m, ROW*COL*sizeof(int));
    hipMallocManaged(&d_v, ROW*sizeof(int));
    hipMallocManaged(&d_res, ROW*sizeof(int));
    // init d_m, d_v on the host
    TestMatrixGenerate(d_m, ROW, COL);
    TestVectorGenerate(d_v, ROW);
    int blockSize = 256;
    mat_vect<<<1, blockSize>>>(d_m, d_v, d_res, ROW, COL);
    
    hipDeviceSynchronize();
    
    for (int i = 0; i < ROW; i++) {
        printf("%d", d_res[i]);
    }
    printf("\n");
    return 0;
}
