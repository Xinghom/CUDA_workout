
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ROW 1000
#define COL 1000

__global__ void mat_vect(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < n && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i];
            printf("cur sum: %d\n", sum);
        }
	printf("col value is : %d\n", col);
	printf("sum value: %d\n", sum);
        c[col] = sum;
    }
}

void TestMatrixGenerate(int* matrix, int row, int col) {
    //printf("Row: %d", row);
    //printf("Col: %d", col);
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            matrix[i * col + j] = (j + 1) + i * col; 
      	    //printf("%d ", matrix[i*col + j]);
        }
        //printf("\n");
    }
}

void TestVectorGenerate(int* vector, int row) {
    for(int i = 0; i < row; i++) {
	vector[i] = i * 2;
 	//printf("%d ", vector[i]);
    }
}


int main() {
    int m[ROW*COL];
    int v[COL];
    int res[COL];

    int *ptr_m, *ptr_v, *ptr_res;
    ptr_m = m;
    ptr_v = v;
    ptr_res = res;
    TestMatrixGenerate(ptr_m, ROW, COL);
    TestVectorGenerate(ptr_v, ROW);
    
    int *d_m, *d_v;
    int *d_res;
    hipMallocManaged(&d_m, ROW*COL*sizeof(int));
    hipMallocManaged(&d_v, ROW*sizeof(int));
    hipMallocManaged(&d_res, ROW*sizeof(int));
    // init d_m, d_v on the host
    //TestMatrixGenerate(d_m, ROW, COL);
    //TestVectorGenerate(d_v, ROW);
    int blockSize = 256;
    mat_vect<<<1, blockSize>>>(d_m, d_v, d_res, ROW, COL);
    
    hipDeviceSynchronize();
    
    // result verify
    printf("the front 10 elements results: \n");
    for (int i = 0; i < 10; i++) {
    	printf("%d ", d_res[i]);
    }   
    
    hipFree(d_m);
    hipFree(d_v);
    hipFree(d_res);
    return 0;
}
