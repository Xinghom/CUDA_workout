
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 20480

//kernel
__global__ void saxpy(float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
    	y[i] += a*x[i];
    }
}

int main(void) {
    float *x, *y, a, *dx, *dy;
    size_t size = N * sizeof(float);
    
    hipMalloc((void **) &dx, size);
    hipMalloc((void **) &dy, size);

    hipMemcpy(dx, x, size, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, size, hipMemcpyHostToDevice);

    saxpy<<<N/256, 256>>>(a, dx, dy);

    hipMemcpy(y, dy, size, hipMemcpyDeviceToHost);
    printf("value of y: %d", y);
    hipFree(dx);
    hipFree(dy);
}
