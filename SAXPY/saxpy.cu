// Adds the elements of 2 arrays with
// a million elements each.


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


//kernel
__global__ void add(int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {    
    int N = 1 << 20; // = 1M elements;
    
    
    /* --- CPU Version ----
    float *x = new float[N];
    float *y = new float[N];

    //init x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    // run add() on CPU (HOST)
    add(N, x, y);

    //Check for errors, == 3.0f?
    float maxError = 0.0f;
    for(int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "MAX Error: " << maxError << std::endl;

    // free mem
    delete [] x;
    delete [] y;
    return 0;
    */
    
    // Allocate Unified Memory -- accessible from CPU or GPU
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // init x and y on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add<<<1,256>>>(N, x, y);

    //wait for GPU
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) 
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max Error: " << maxError << std::endl;

    // free Mem
    hipFree(x);
    hipFree(y);
    return 0;




    // cudaMalloc((void **) &da, size);
    // cudaMalloc((void **) &db, size);
    // cudaMalloc((void **) &dc, size);

    // //setup input values
    // a = 2;
    // b = 7;

    // cudaMemcpy(da, &a, size, cudaMemcpyHostToDevice);
    // cudaMemcpy(db, &b, size, cudaMemcpyHostToDevice);

    // add<<<1, 1>>>(da, db, dc);

    // cudaMemcpy(&c, dc, size, cudaMemcpyDeviceToHost);

    // printf("Value of c: %d", c);
    // cudaFree(da);
    // cudaFree(db);
    // cudaFree(dc);
    // return 0;
}
