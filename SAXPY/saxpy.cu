
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 2048

//kernel
__global__ void saxpy(float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
    	y[i] += a*x[i];
    }
}

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}
int main(void) {    
    int a, b, c; //host copies
    int *da, *db, *dc;
    int size = sizeof(int);



    hipMalloc((void **) &da, size);
    hipMalloc((void **) &db, size);
    hipMalloc((void **) &dc, size);

    //setup input values
    a = 2;
    b = 7;

    hipMemcpy(da, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, &b, size, hipMemcpyHostToDevice);

    add<<<1, 1>>>(da, db, dc);

    hipMemcpy(&c, dc, size, hipMemcpyDeviceToHost);

    printf("Value of c: %d", c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}
