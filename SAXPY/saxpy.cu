
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 20480

//kernel
__global__ void saxpy(float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
    	y[i] += a*x[i];
    }
}

int main(void) {
    float *x, *y, a, *dx, *dy;
    size_t size = N * sizeof(float);
}
