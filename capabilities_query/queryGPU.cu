/*
To query the number of CUDA-capable GPUs
in a host and the capabilities of each GPU. 
Run it on the Hummingbird GPU node and 
report the results.
*/

#include<stdio.h>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("GPU Num: %d\n", i);
        printf("  - GPU Name: %s\n", prop.name);
        printf("  - Mem Speed: %d\n", prop.memoryClockRate);
        printf("  - Mem Bus Width (bits): %d\n", prop.memoryBusWidth);
        float MemPeakBandWidth = 2.0 * prop.memoryClockRate * (prop.memoryBusWidth/8) / 1.0e6;
        printf("  - Mem Peak Bandwidth(GB/s): %f\n", MemPeakBandWidth);
        printf("*** Compute Capability: %d\n", prop.major);
    }
    return 0;
}
